#include "hip/hip_runtime.h"
// #define _GNU_SOURCE
#include "cracker-gpu.h"

#include <openssl/md5.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <pthread.h>
#include <sys/time.h>
#include <stdbool.h>


#include "util.h"


// How many characters do we have to search through
#define ALPHABET_SIZE 26

//TODO: maybe have an array of all the characters in the alphabet?

// Each board will be a block
#define THREADS_PER_BLOCK (ALPHABET_SIZE * ALPHABET_SIZE)
#define PASSWORDS_PER_THREAD (ALPHABET_SIZE * ALPHABET_SIZE)
#define PASSWORDS_PER_BLOCK (PASSWORDS_PER_THREAD * THREADS_PER_BLOCK)

#define SEARCH_SPACE_SIZE (pow(26, PASSWORD_LENGTH))

#define TRUE 1
#define FALSE 0

/************************* MD5 *************************/
// Look into: https://github.com/VladX/md5-bruteforcer/blob/master/gpu.cu
/*
 * Derived from the RSA Data Security, Inc. MD5 Message-Digest Algorithm
 * and modified slightly to be functionally identical but condensed into control structures.
 * From Zunawe at https://github.com/Zunawe/md5-c/blob/main/md5.c
 */

/*
 * Constants defined by the MD5 algorithm
 */
#define A 0x67452301
#define B 0xefcdab89
#define C 0x98badcfe
#define D 0x10325476

typedef struct{
    uint64_t size;        // Size of input in bytes
    uint32_t buffer[4];   // Current accumulation of hash
    uint8_t input[64];    // Input to be used in the next step
    uint8_t digest[16];   // Result of algorithm
}MD5Context;

__device__ void md5Init(MD5Context *ctx);
__device__ void md5Update(MD5Context *ctx, uint8_t *input, size_t input_len);
__device__ void md5Finalize(MD5Context *ctx);
__device__ void md5Step(uint32_t *buffer, uint32_t *input);
 
__device__ void md5String(char *input, uint8_t *result);

__constant__ uint32_t S[64];
__constant__ uint32_t K[64];
__constant__ uint8_t PADDING[64];

static uint32_t cpuS[] = {7, 12, 17, 22, 7, 12, 17, 22, 7, 12, 17, 22, 7, 12, 17, 22,
                       5,  9, 14, 20, 5,  9, 14, 20, 5,  9, 14, 20, 5,  9, 14, 20,
                       4, 11, 16, 23, 4, 11, 16, 23, 4, 11, 16, 23, 4, 11, 16, 23,
                       6, 10, 15, 21, 6, 10, 15, 21, 6, 10, 15, 21, 6, 10, 15, 21};



static uint32_t cpuK[] = {0xd76aa478, 0xe8c7b756, 0x242070db, 0xc1bdceee,
                       0xf57c0faf, 0x4787c62a, 0xa8304613, 0xfd469501,
                       0x698098d8, 0x8b44f7af, 0xffff5bb1, 0x895cd7be,
                       0x6b901122, 0xfd987193, 0xa679438e, 0x49b40821,
                       0xf61e2562, 0xc040b340, 0x265e5a51, 0xe9b6c7aa,
                       0xd62f105d, 0x02441453, 0xd8a1e681, 0xe7d3fbc8,
                       0x21e1cde6, 0xc33707d6, 0xf4d50d87, 0x455a14ed,
                       0xa9e3e905, 0xfcefa3f8, 0x676f02d9, 0x8d2a4c8a,
                       0xfffa3942, 0x8771f681, 0x6d9d6122, 0xfde5380c,
                       0xa4beea44, 0x4bdecfa9, 0xf6bb4b60, 0xbebfbc70,
                       0x289b7ec6, 0xeaa127fa, 0xd4ef3085, 0x04881d05,
                       0xd9d4d039, 0xe6db99e5, 0x1fa27cf8, 0xc4ac5665,
                       0xf4292244, 0x432aff97, 0xab9423a7, 0xfc93a039,
                       0x655b59c3, 0x8f0ccc92, 0xffeff47d, 0x85845dd1,
                       0x6fa87e4f, 0xfe2ce6e0, 0xa3014314, 0x4e0811a1,
                       0xf7537e82, 0xbd3af235, 0x2ad7d2bb, 0xeb86d391};

/*
 * Padding used to make the size (in bits) of the input congruent to 448 mod 512
 */
static uint8_t cpuPADDING[] = {0x80, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00,
                            0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00,
                            0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00,
                            0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00,
                            0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00,
                            0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00,
                            0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00,
                            0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00};

/*
 * Bit-manipulation functions defined by the MD5 algorithm
 */
#define F(X, Y, Z) ((X & Y) | (~X & Z))
#define G(X, Y, Z) ((X & Z) | (Y & ~Z))
#define H(X, Y, Z) (X ^ Y ^ Z)
#define I(X, Y, Z) (Y ^ (X | ~Z))

/*
 * Rotates a 32-bit word left by n bits
 */
__device__ uint32_t rotateLeft(uint32_t x, uint32_t n){
    return (x << n) | (x >> (32 - n));
}


/*
 * Initialize a context
 */
__device__ void md5Init(MD5Context *ctx){
    ctx->size = (uint64_t)0;

    ctx->buffer[0] = (uint32_t)A;
    ctx->buffer[1] = (uint32_t)B;
    ctx->buffer[2] = (uint32_t)C;
    ctx->buffer[3] = (uint32_t)D;
}

/*
 * Add some amount of input to the context
 *
 * If the input fills out a block of 512 bits, apply the algorithm (md5Step)
 * and save the result in the buffer. Also updates the overall size.
 */
__device__ void md5Update(MD5Context *ctx, uint8_t *input_buffer, size_t input_len){
    uint32_t input[16];
    unsigned int offset = ctx->size % 64;
    ctx->size += (uint64_t)input_len;

    // Copy each byte in input_buffer into the next space in our context input
    for(unsigned int i = 0; i < input_len; ++i){
        ctx->input[offset++] = (uint8_t)*(input_buffer + i);

        // If we've filled our context input, copy it into our local array input
        // then reset the offset to 0 and fill in a new buffer.
        // Every time we fill out a chunk, we run it through the algorithm
        // to enable some back and forth between cpu and i/o
        if(offset % 64 == 0){
            for(unsigned int j = 0; j < 16; ++j){
                // Convert to little-endian
                // The local variable `input` our 512-bit chunk separated into 32-bit words
                // we can use in calculations
                input[j] = (uint32_t)(ctx->input[(j * 4) + 3]) << 24 |
                           (uint32_t)(ctx->input[(j * 4) + 2]) << 16 |
                           (uint32_t)(ctx->input[(j * 4) + 1]) <<  8 |
                           (uint32_t)(ctx->input[(j * 4)]);
            }
            md5Step(ctx->buffer, input);
            offset = 0;
        }
    }
}

/*
 * Pad the current input to get to 448 bytes, append the size in bits to the very end,
 * and save the result of the final iteration into digest.
 */
__device__ void md5Finalize(MD5Context *ctx){
    uint32_t input[16];
    unsigned int offset = ctx->size % 64;
    unsigned int padding_length = offset < 56 ? 56 - offset : (56 + 64) - offset;

    // Fill in the padding and undo the changes to size that resulted from the update
    md5Update(ctx, PADDING, padding_length);
    ctx->size -= (uint64_t)padding_length;

    // Do a final update (internal to this function)
    // Last two 32-bit words are the two halves of the size (converted from bytes to bits)
    for(unsigned int j = 0; j < 14; ++j){
        input[j] = (uint32_t)(ctx->input[(j * 4) + 3]) << 24 |
                   (uint32_t)(ctx->input[(j * 4) + 2]) << 16 |
                   (uint32_t)(ctx->input[(j * 4) + 1]) <<  8 |
                   (uint32_t)(ctx->input[(j * 4)]);
    }
    input[14] = (uint32_t)(ctx->size * 8);
    input[15] = (uint32_t)((ctx->size * 8) >> 32);

    md5Step(ctx->buffer, input);

    // Move the result into digest (convert from little-endian)
    for(unsigned int i = 0; i < 4; ++i){
        ctx->digest[(i * 4) + 0] = (uint8_t)((ctx->buffer[i] & 0x000000FF));
        ctx->digest[(i * 4) + 1] = (uint8_t)((ctx->buffer[i] & 0x0000FF00) >>  8);
        ctx->digest[(i * 4) + 2] = (uint8_t)((ctx->buffer[i] & 0x00FF0000) >> 16);
        ctx->digest[(i * 4) + 3] = (uint8_t)((ctx->buffer[i] & 0xFF000000) >> 24);
    }
}

/*
 * Step on 512 bits of input with the main MD5 algorithm.
 */
__device__ void md5Step(uint32_t *buffer, uint32_t *input){
    uint32_t AA = buffer[0];
    uint32_t BB = buffer[1];
    uint32_t CC = buffer[2];
    uint32_t DD = buffer[3];

    uint32_t E;

    unsigned int j;

    for(unsigned int i = 0; i < 64; ++i){
        switch(i / 16){
            case 0:
                E = F(BB, CC, DD);
                j = i;
                break;
            case 1:
                E = G(BB, CC, DD);
                j = ((i * 5) + 1) % 16;
                break;
            case 2:
                E = H(BB, CC, DD);
                j = ((i * 3) + 5) % 16;
                break;
            default:
                E = I(BB, CC, DD);
                j = (i * 7) % 16;
                break;
        }

        uint32_t temp = DD;
        DD = CC;
        CC = BB;
        BB = BB + rotateLeft(AA + E + K[i] + input[j], S[i]);
        AA = temp;
    }

    buffer[0] += AA;
    buffer[1] += BB;
    buffer[2] += CC;
    buffer[3] += DD;
}

/*
 * Functions that run the algorithm on the provided input and put the digest into result.
 * result should be able to store 16 bytes.
 */
__device__ void md5String(char *input, size_t length, uint8_t *result){
    MD5Context ctx;
    md5Init(&ctx);
    md5Update(&ctx, (uint8_t *)input, length);
    md5Finalize(&ctx);

    memcpy(result, ctx.digest, 16);
}


__device__ bool hashcmp(uint8_t * one, uint8_t * theOther){
  for(int i = 0; i < MD5_DIGEST_LENGTH; i++){
    if(one[i] != theOther[i]){
      return false;
    }
  }
  return true;
}

__device__ void pwdcpy(char * from, char * to){
  for(int i = 0; i < PASSWORD_LENGTH; i++){
    to[i] = from[i];
  }
}

void usrnmcpy(char * from, char * to){
  for(int i = 0; i < PASSWORD_LENGTH; i++){
    to[i] = from[i];
  }
}




/********************* Parts B & C ************************/



__constant__ int numPasswordsGPU;
__constant__ int numUsersGPU;
__constant__ int offsetGPU;
int numPasswords;





/**
 * Add a password to a password set
 * \param passwords   A pointer to a password set initialized with the function above.
 * \param username    The name of the user being added. The memory that holds this string's
 *                    characters will be reused, so if you keep a copy you must duplicate the
 *                    string. I recommend calling strdup().
 * \param password_hash   An array of MD5_DIGEST_LENGTH bytes that holds the hash of this user's
 *                        password. The memory that holds this array will be reused, so you must
 *                        make a copy of this value if you retain it in your data structure.
 */
// void add_password(password_set_t* passwords, char* username, uint8_t* password_hash) {
//   // Malloc space for the node
//   password_set_node_t* node = (password_set_node_t*) malloc(sizeof(password_set_node_t));

//   // Then assign all the fields
//   node->username = strdup(username);
//   memcpy(node->hashed_password, password_hash, MD5_DIGEST_LENGTH);
//   node->next = passwords->buckets[password_hash[0] & numBucketsAndMask];
//   node->prev = NULL;

//   // And link the next's node's prev to this ones
//   if(node->next != NULL){
//     node->next->prev = node;
//   }

//   // Then update the bucket reference
//   passwords->buckets[password_hash[0] & numBucketsAndMask] = node;

//   // And the # of passwords
//   // passwords->numPasswords++;
// }

// This is the fuction that runs on each thread
__global__ void cracker_thread(password_set_node_t* passwords){
  // printf("test");
  // And declare local
  uint8_t candidate_hash[MD5_DIGEST_LENGTH]; //< This will hold the hash of the candidate password

  // Same as the individual thread, but now we have each start on offset, and inc by the # of threads
  char candidate_passwd[] = "aaaaaaa";
  candidate_passwd[0]+= offsetGPU;
  candidate_passwd[2]+= threadIdx.x;
  candidate_passwd[3]+= threadIdx.y;
  // candidate_passwd[2]+= threadIdx.z;
  candidate_passwd[4]+= blockIdx.x % 26;
  candidate_passwd[5]+= (blockIdx.x / 26) % 26;
  candidate_passwd[6]+= ((blockIdx.x / 26) / 26) % 26;
  // candidate_passwd[6]+= (((blockIdx.x / 26) / 26) / 26) % 26;
  // MD5((unsigned char*)candidate_passwd, PASSWORD_LENGTH, candidate_hash); //< Do the hash (this is the slowest part of this implementation)
  int hash_index;

  for(int i = 0; i < ALPHABET_SIZE; i++){
    for(int j = 0; j < ALPHABET_SIZE; j++){
      md5String(candidate_passwd, PASSWORD_LENGTH, candidate_hash);

  // Get the bucket corresponding to the hash
  // // password = passwords->buckets[candidate_hash[0] & numBucketsAndMask];
  // if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0 && blockIdx.x == 0){
  //   // passwords[0].username;
  //   printf("%s %x", candidate_passwd, passwords[0].hashed_password);
  //   // cuPrintf("test!!!");
  // }
      hash_index = (candidate_passwd[0] & numBucketsAndMask);

      // Now check if the hash of the candidate password matches any of the hashs in the bucket, 
      // going along till we get to an empty one. Since they needed to be all sent together, an array was best
      while(passwords[hash_index].hashed_password[0] != 0){
        if(hashcmp(candidate_hash, passwords[hash_index].hashed_password)){
          // hipMemcpy(&(passwords[i].solved_password), candidate_passwd, sizeof(char) * PASSWORD_LENGTH, hipMemcpyDeviceToDevice);
          pwdcpy(candidate_passwd, passwords[hash_index].solved_password);
          // printf("%s", candidate_passwd);
          break;
        }
        hash_index = (hash_index + 1) % (numBucketsAndMask + 1);
    }
      // while(int i = 0; i < numPasswordsGPU; i++){
      //   // if(memcmp(candidate_hash, &(passwords[i].hashed_password), MD5_DIGEST_LENGTH) == 0) {
        
      // }
      candidate_passwd[0] += numUsersGPU;
    }
    candidate_passwd[0] = 'a';
    candidate_passwd[1]++;
  }
  // Potential TODO: Add check somewhere if we've cracked all passwords? This would be tough among all 
  //                  the different computers and threads.
}

/**
 * Crack all of the passwords in a set of passwords. The function should print the username
 * and cracked password for each user listed in passwords, separated by a space character.
 * Complete this implementation for part B of the lab.
 *
 * \returns The number of passwords cracked in the list
 */
void crack_password_list_num(password_set_node_t* argsPasswords, size_t numPasswordsArg, int index, int numUsers) {
  // Change the buffer so we don't waste time on constant system calls and context switches
  // char buffer[2048];
  // setvbuf(stdout, buffer, _IOFBF, 2048);
  numPasswords = numPasswordsArg;


  // if (hipMalloc(&K, sizeof(uint32_t) * 64) != hipSuccess) {
  //   fprintf(stderr, "Failed to allocate K array on GPU\n");
  //   exit(2);
  // }
  if (hipMemcpyToSymbol(HIP_SYMBOL(K), cpuK, sizeof(uint32_t) * 64, 0, hipMemcpyHostToDevice) !=
      hipSuccess) {
    fprintf(stderr, "Failed to copy K to the GPU\n");
  }

  if (hipMemcpyToSymbol(HIP_SYMBOL(S), cpuS, sizeof(uint32_t) * 64, 0, hipMemcpyHostToDevice) !=
      hipSuccess) {
    fprintf(stderr, "Failed to copy S to the GPU\n");
  }

  if (hipMemcpyToSymbol(HIP_SYMBOL(numUsersGPU), &numUsers, sizeof(int), 0, hipMemcpyHostToDevice) !=
      hipSuccess) {
    fprintf(stderr, "Failed to copy numUsers to the GPU\n");
  }

  if (hipMemcpyToSymbol(HIP_SYMBOL(offsetGPU), &index, sizeof(int), 0, hipMemcpyHostToDevice) !=
      hipSuccess) {
    fprintf(stderr, "Failed to copy index to the GPU\n");
  }

  if (hipMemcpyToSymbol(HIP_SYMBOL(PADDING), cpuPADDING, sizeof(uint8_t) * 64, 0, hipMemcpyHostToDevice) !=
      hipSuccess) {
    fprintf(stderr, "Failed to copy PADDING to the GPU\n");
  }

  // Just do array for now will need to change to accomodate HASH
  password_set_node_t* GPUpasswords;

  // Allocate space for the boards on the GPU
  if (hipMalloc(&GPUpasswords, sizeof(password_set_node_t) * (numBucketsAndMask + 1)) != hipSuccess) {
    fprintf(stderr, "Failed to allocate passwords array on GPU\n");
    exit(2);
  }

  // Copy the cpu's x array to the gpu with hipMemcpy
  if (hipMemcpy(GPUpasswords, argsPasswords, sizeof(password_set_node_t) * (numBucketsAndMask + 1), hipMemcpyHostToDevice) !=
      hipSuccess) {
    fprintf(stderr, "Failed to copy password to the GPU\n");
  }

  
  // dim3 layout(ALPHABET_SIZE, ALPHABET_SIZE, ALPHABET_SIZE);
  size_t blocks = (SEARCH_SPACE_SIZE + PASSWORDS_PER_BLOCK - 1) / PASSWORDS_PER_BLOCK;
  cracker_thread<<<blocks, dim3(26, 26)>>>(GPUpasswords); // Actually run the solver on each thread
  // cracker_thread<<<blocks, layout>>>(GPUpasswords); // Actually run the solver on each thread

  // Wait for all the threads to finish
  if (hipDeviceSynchronize() != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(hipPeekAtLastError()));
  }

  // Copy the solved array back from the gpu to the cpu
  if(hipMemcpy(argsPasswords, GPUpasswords, sizeof(password_set_node_t) * (numBucketsAndMask + 1), hipMemcpyDeviceToHost) != hipSuccess) {
    fprintf(stderr, "Failed to copy back from the GPU\n");
  }

  // Free the board memory on the GPU
  hipFree(GPUpasswords);
  hipFree(K);
  hipFree(S);
  hipFree(PADDING);

  for(int i = 0; i < (numBucketsAndMask + 1); i++){
    if(argsPasswords[i].hashed_password[0] != 0){
      printf("%s %.*s\n", argsPasswords[i].username, PASSWORD_LENGTH, argsPasswords[i].solved_password);
    }
  }
}


void crack_password_list(password_set_node_t* passwords) {
  crack_password_list_num(passwords, 256, 1, 1);
}

/******************** Provided Code ***********************/

/**
 * Convert a string representation of an MD5 hash to a sequence
 * of bytes. The input md5_string must be 32 characters long, and
 * the output buffer bytes must have room for MD5_DIGEST_LENGTH
 * bytes.
 *
 * \param md5_string  The md5 string representation
 * \param bytes       The destination buffer for the converted md5 hash
 * \returns           0 on success, -1 otherwise
 */
int md5_string_to_bytes(const char* md5_string, uint8_t* bytes) {
  // Check for a valid MD5 string
  if(strlen(md5_string) != 2 * MD5_DIGEST_LENGTH) return -1;

  // Start our "cursor" at the start of the string
  const char* pos = md5_string;

  // Loop until we've read enough bytes
  for(size_t i=0; i<MD5_DIGEST_LENGTH; i++) {
    // Read one byte (two characters)
    int rc = sscanf(pos, "%2hhx", &bytes[i]);
    if(rc != 1) return -1;

    // Move the "cursor" to the next hexadecimal byte
    pos += 2;
  }

  return 0;
}

void print_usage(const char* exec_name) {
  fprintf(stderr, "Usage:\n");
  fprintf(stderr, "  %s single <MD5 hash>\n", exec_name);
  fprintf(stderr, "  %s list <password file name>\n", exec_name);
}

// int main(int argc, char** argv) {
//   if(argc != 3) {
//     print_usage(argv[0]);
//     exit(1);
//   }

//   if(strcmp(argv[1], "list") == 0) {
//     // Make and initialize a password set
//     password_set_node_t* passwords = NULL;
//     // init_password_set(&passwords);

//     // Open the password file
//     FILE* password_file = fopen(argv[2], "r");
//     if(password_file == NULL) {
//       perror("opening password file");
//       exit(2);
//     }

//     // Read until we hit the end of the file
//     while(!feof(password_file)) {
//       // Make space to hold the username
//       char username[MAX_USERNAME_LENGTH];

//       // Make space to hold the MD5 string
//       char md5_string[MD5_DIGEST_LENGTH * 2 + 1];

//       // Make space to hold the MD5 bytes
//       uint8_t password_hash[MD5_DIGEST_LENGTH];

//       // Try to read. The space in the format string is required to eat the newline
//       if(fscanf(password_file, "%s %s ", username, md5_string) != 2) {
//         fprintf(stderr, "Error reading password file: malformed line\n");
//         exit(2);
//       }

//       // Convert the MD5 string to MD5 bytes in our new node
//       if(md5_string_to_bytes(md5_string, password_hash) != 0) {
//         fprintf(stderr, "Error reading MD5\n");
//         exit(2);
//       }

//       // Add the password to the password set
//       // add_password(&passwords, username, password_hash);
//       add_password_array(&passwords, username, password_hash);
//     }

//     // Now run the password list cracker
//     crack_password_list(passwords);

//   } else {
//     print_usage(argv[0]);
//     exit(1);
//   }

//   return 0;
// }